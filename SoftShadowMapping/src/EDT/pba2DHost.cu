#include "hip/hip_runtime.h"
/*
Author: Cao Thanh Tung
Date: 21/01/2010

File Name: pba2DHost.cu

===============================================================================

Copyright (c) 2010, School of Computing, National University of Singapore. 
All rights reserved.

Project homepage: http://www.comp.nus.edu.sg/~tants/pba.html

If you use PBA and you like it or have comments on its usefulness etc., we 
would love to hear from you at <tants@comp.nus.edu.sg>. You may share with us
your experience and any possibilities that we may improve the work/code.

===============================================================================

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

Redistributions of source code must retain the above copyright notice, this list of
conditions and the following disclaimer. Redistributions in binary form must reproduce
the above copyright notice, this list of conditions and the following disclaimer
in the documentation and/or other materials provided with the distribution. 

Neither the name of the National University of University nor the names of its contributors
may be used to endorse or promote products derived from this software without specific
prior written permission from the National University of Singapore. 

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO THE IMPLIED WARRANTIES 
OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
TO, PROCUREMENT OF SUBSTITUTE  GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
DAMAGE.

*/

#include <hip/device_functions.h>

#include "EDT\pba2D.h"

// Parameters for CUDA kernel executions
#define BLOCKX		16
#define BLOCKY		16
#define BLOCKSIZE	64
#define TILE_DIM	16
#define BLOCK_ROWS	8

/****** Global Variables *******/
short2 **pbaTextures;       // Two textures used to compute 2D Voronoi Diagram
short2 *pbaTransposed;

int pbaMemSize;             // Size (in bytes) of a texture
int pbaTexWidth;             // Texture size (squared texture)
int pbaTexHeight;             // Texture size (squared texture)

texture<short2> pbaTexColor; 
texture<short2> pbaTexLinks; 
texture<float4, hipTextureType2D, hipReadModeElementType> pbaImage;
texture<float4, hipTextureType2D, hipReadModeElementType> pbaGlobalPosition;
hipArray* arrayDevice[2];
float* GPUNormalizedEDTImage;

/********* Kernels ********/
#include "EDT\pba2DKernel.h"

///////////////////////////////////////////////////////////////////////////
//
// Initialize necessary memory for 2D Voronoi Diagram computation
// - textureSize: The size of the Discrete Voronoi Diagram (width = height)
//
///////////////////////////////////////////////////////////////////////////
void pba2DInitialization(int textureWidth, int textureHeight)
{
    pbaTexWidth = textureWidth;
	pbaTexHeight = textureHeight; 
    pbaMemSize = pbaTexWidth * pbaTexHeight * sizeof(short2); 

    pbaTextures = (short2 **) malloc(2 * sizeof(short2 *)); 

    // Allocate 2 textures
    hipMalloc((void **) &pbaTextures[0], pbaMemSize); 
    hipMalloc((void **) &pbaTextures[1], pbaMemSize); 
	hipMalloc((void **) &pbaTransposed, pbaMemSize);
	hipMalloc(&GPUNormalizedEDTImage, pbaTexWidth * pbaTexHeight * 4 * sizeof(float));
	
}

///////////////////////////////////////////////////////////////////////////
//
// Deallocate all allocated memory
//
///////////////////////////////////////////////////////////////////////////
void pba2DDeinitialization()
{
    hipFree(pbaTextures[0]); 
    hipFree(pbaTextures[1]); 
	hipFree(pbaTransposed);
	hipFree(GPUNormalizedEDTImage);
    free(pbaTextures); 
}

// Copy input to GPU 
void pba2DInitializeInput(short *input)
{
    hipMemcpy(pbaTextures[0], input, pbaMemSize, hipMemcpyHostToDevice); 
}

void pba2DInitializeInput(float siteValue)
{
	initializeInput<<<(int)ceilf((pbaTexWidth * pbaTexHeight)/512), 512>>>(pbaTextures[0], pbaTexHeight, pbaTexWidth, siteValue);
}

// In-place transpose a squared texture. 
// Block orders are modified to optimize memory access. 
// Point coordinates are also swapped. 
void pba2DTranspose(short2 *output, short2 *input, int pbaTempWidth, int pbaTempHeight)
{
    dim3 block(TILE_DIM, BLOCK_ROWS); 
    dim3 grid(pbaTempWidth / TILE_DIM, pbaTempHeight / TILE_DIM); 
	//hipBindTexture(0, pbaTexColor, pbaTextures[1]); 
    kernelTranspose<<< grid, block>>>(output, input, pbaTempWidth, pbaTempHeight); 
	//kernelTranspose<<<grid, block>>>(pbaTextures[1], pbaTempWidth);
	//hipUnbindTexture(pbaTexColor); 

}

// Phase 1 of PBA. m1 must divides texture size
void pba2DPhase1(int m1) 
{
    dim3 block = dim3(BLOCKSIZE);   
    dim3 grid = dim3(pbaTexWidth / block.x, m1); 

    // Flood vertically in their own bands
    hipBindTexture(0, pbaTexColor, pbaTextures[0]); 
    kernelFloodDown<<< grid, block>>>(pbaTextures[1], pbaTexWidth, pbaTexHeight / m1); 
    hipUnbindTexture(pbaTexColor); 

    hipBindTexture(0, pbaTexColor, pbaTextures[1]); 
    kernelFloodUp<<< grid, block>>>(pbaTextures[1], pbaTexWidth, pbaTexHeight / m1); 

    // Passing information between bands
    grid = dim3(pbaTexWidth / block.x, m1); 
    kernelPropagateInterband<<< grid, block>>>(pbaTextures[0], pbaTexWidth, pbaTexHeight / m1); 

    hipBindTexture(0, pbaTexLinks, pbaTextures[0]); 
    kernelUpdateVertical<<< grid, block>>>(pbaTextures[1], pbaTexWidth, m1, pbaTexHeight / m1); 
    hipUnbindTexture(pbaTexLinks); 
    hipUnbindTexture(pbaTexColor); 
}

// Phase 2 of PBA. m2 must divides texture size
void pba2DPhase2(int m2) 
{
    // Compute proximate points locally in each band
    dim3 block = dim3(BLOCKSIZE);   
    dim3 grid = dim3(pbaTexHeight / block.x, m2); 
    hipBindTexture(0, pbaTexColor, pbaTransposed); 
    //hipBindTexture(0, pbaTexColor, pbaTextures[1]);
	kernelProximatePoints<<< grid, block>>>(pbaTextures[0], pbaTexHeight, pbaTexWidth / m2); 
	
    hipBindTexture(0, pbaTexLinks, pbaTextures[0]); 
    kernelCreateForwardPointers<<< grid, block>>>(pbaTextures[0], pbaTexHeight, pbaTexWidth / m2); 
	
    // Repeatly merging two bands into one
    for (int noBand = m2; noBand > 1; noBand /= 2) {
        grid = dim3(pbaTexHeight / block.x, noBand / 2); 
        kernelMergeBands<<< grid, block>>>(pbaTextures[0], pbaTexHeight, pbaTexWidth / noBand); 
    }
	
    // Replace the forward link with the X coordinate of the seed to remove
    // the need of looking at the other texture. We need it for coloring.
    grid = dim3(pbaTexWidth / block.x, pbaTexWidth); 
    kernelDoubleToSingleList<<< grid, block>>>(pbaTextures[0], pbaTexHeight); 
	
    hipUnbindTexture(pbaTexLinks); 
    hipUnbindTexture(pbaTexColor); 
	
}

// Phase 3 of PBA. m3 must divides texture size
void pba2DPhase3(int m3) 
{
    dim3 block = dim3(BLOCKSIZE / m3, m3); 
    dim3 grid = dim3(pbaTexHeight / block.x); 
    //dim3 block = dim3(BLOCKSIZE);   
    //dim3 grid = dim3(pbaTexHeight / block.x, m3); 

	hipBindTexture(0, pbaTexColor, pbaTextures[0]); 
    kernelColor<<< grid, block>>>(pbaTransposed, pbaTexHeight, pbaTexWidth); 
    //kernelColor<<< grid, block>>>(pbaTextures[1], pbaTexHeight);
	hipUnbindTexture(pbaTexColor); 
}

void pba2DCompute(int floodBand, int maurerBand, int colorBand)
{
    // Vertical sweep
    pba2DPhase1(floodBand); 

    pba2DTranspose(pbaTransposed, pbaTextures[1], pbaTexWidth, pbaTexHeight); 
	
    // Horizontal coloring
    pba2DPhase2(maurerBand); 

    // Color the rows. 
    pba2DPhase3(colorBand); 
	
    pba2DTranspose(pbaTextures[1], pbaTransposed, pbaTexHeight, pbaTexWidth);    
	
}

void pba2DNormalizeEDT(float shadowIntensity) {

	pbaNormalizeDistanceTransform<<<(int)ceilf(pbaTexWidth * pbaTexHeight/512), 512>>>(pbaTextures[1], GPUNormalizedEDTImage, pbaTexWidth, shadowIntensity);
	hipMemcpyToArray(arrayDevice[0], 0, 0, GPUNormalizedEDTImage, pbaTexWidth * pbaTexHeight * 4 * sizeof(float), hipMemcpyDeviceToDevice);

}

void pba2DEDT(float shadowIntensity) {

	pba2DEDTKernel<<<(int)ceilf(pbaTexWidth * pbaTexHeight/512), 512>>>(pbaTextures[1], GPUNormalizedEDTImage, pbaTexWidth, shadowIntensity);
	hipMemcpyToArray(arrayDevice[0], 0, 0, GPUNormalizedEDTImage, pbaTexWidth * pbaTexHeight * 4 * sizeof(float), hipMemcpyDeviceToDevice);

}

// Compute 2D Voronoi diagram
// Input: a 2D texture. Each pixel is represented as two "short" integer. 
//    For each site at (x, y), the pixel at coordinate (x, y) should contain 
//    the pair (x, y). Pixels that are not sites should contain the pair (MARKER, MARKER)
// See original paper for the effect of the three parameters: 
//    phase1Band, phase2Band, phase3Band
// Parameters must divide textureSize
void pba2DVoronoiDiagram(int floodBand, int maurerBand, int colorBand, float shadowIntensity) 
{

	//Initialize sites
	pba2DInitializeInput(0.0);

	// Compute umbra EDT
    pba2DCompute(floodBand, maurerBand, colorBand); 

	// Copy back to an umbra EDT image
	pba2DEDT(shadowIntensity);

	/*   
	// Initialize umbra sites
    pba2DInitializeInput(0.0); 

    // Compute umbra EDT
    pba2DCompute(floodBand, maurerBand, colorBand); 

	// Copy back to an umbra EDT image
	pba2DEDT(shadowIntensity);

	// Initialize lit sites
	pba2DInitializeInput(1.0);

	// Compute lit EDT
    pba2DCompute(floodBand, maurerBand, colorBand); 

	// Normalize EDT
	pba2DNormalizeEDT(shadowIntensity);
	*/

}

void pbaCudaBindTexture(hipGraphicsResource_t *resource) {

	hipGraphicsMapResources(2, resource);

	hipGraphicsSubResourceGetMappedArray(&arrayDevice[0], resource[0], 0, 0);
	hipBindTextureToArray(pbaImage, arrayDevice[0] );

	hipGraphicsSubResourceGetMappedArray(&arrayDevice[1], resource[1], 0, 0);
	hipBindTextureToArray(pbaGlobalPosition, arrayDevice[1] );

}

void pbaCudaUnbindTexture(hipGraphicsResource_t *resource) {

	hipUnbindTexture(pbaImage);
	hipUnbindTexture(pbaGlobalPosition);
	hipGraphicsUnmapResources(2, resource);

}