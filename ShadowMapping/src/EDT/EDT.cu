#include "hip/hip_runtime.h"
#include "EDT\EDT.h"

#define BLOCK_DIM 1024
texture<float4, hipTextureType2D, hipReadModeElementType> image;
hipArray* arrayDevice2;
	
__device__ float GPUComputeEuclideanDistance(int sitePixel, int pixel, int cols) {

	int sx = sitePixel % cols;
	int sy = sitePixel / cols;
	int px = pixel % cols;
	int py = pixel / cols;
	return sqrtf((sx - px) * (sx - px) + (sy - py) * (sy - py));

}

__device__ bool GPUHasDomination(int a, int b, int c, int column, int cols) {

	float u, v;
	//p(i, u)
	int ax = a % cols;
	int ay = a / cols;
	int bx = b % cols;
	int by = b / cols;
	float mx = (float)(ax + bx) / 2;
	float my = (float)(ay + by) / 2;
	if(bx == ax) {
		u = my;
	} else if(by == ay) {
		u = my;
	} else {
		float m1 = (float)(by - ay) / (float)(bx - ax);
		float m2 = -1/m1;
		u = m2 * (column - mx) + my;
	}

	//q(i, v)
	int cx = c % cols;
	int cy = c / cols;
	mx = (float)(bx + cx) / 2;
	my = (float)(by + cy) / 2;
	if(cx == bx) {
		v = my;
	} else if(cy == by) {
		v = my;
	} else {
		float m1 = (float)(cy - by) / (float)(cx - bx);
		float m2 = -1/m1;
		v = m2 * (column - mx) + my;
	}
	
	if(u > v) return true;
	else return false;

}

__device__ int GPUDetectSkeleton(int x, int y, int p, int q, int nx, int ny, int np, int nq, int gamma, int cols) {

	int dif = (int)((nx - np) * (nx - np) + (ny - nq) * (ny - nq));
	if(dif > gamma && dif > sqrtf((x-nx+p-np) * (x-nx+p-np) + (y-ny+q-nq) * (y-ny+q-nq))) {
		int innerProduct = (nx-np) * (nx+np-x-p) + (ny-nq) * (ny+nq-y-q);
		if(innerProduct >= 0) return y * cols + x;
		else return q * cols + p;
	}
	return -1;

}

__global__ void clearStructure(int *structure) {

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	structure[id] = -1;

}

__global__ void computeSelfNearestSite(int *nearestSite, float siteValue, int rows, int cols) {
	
	int pixel = blockIdx.x * blockDim.x + threadIdx.x;
	if(tex2D(image, pixel % cols, pixel / cols).x == siteValue) nearestSite[pixel] = pixel;
	else nearestSite[pixel] = -1;

}

__global__ void computeNearestSiteInRow(int *nearestSite, float siteValue, int rows, int cols, int bandSize) {
	
	int globalId = blockIdx.x * blockDim.x + threadIdx.x;
	int x = (globalId / cols) * bandSize;
	int y = globalId % cols;
	//int x = blockIdx.x * bandSize;
	//int y = threadIdx.x;

	#pragma unroll
	for(int xp = x; xp < x + bandSize; xp++) {
	  
		int pixel = y * cols + xp;
		if(tex2D(image, pixel % cols, pixel / cols).x == siteValue) {

			#pragma unroll
			for(int xs = xp + 1; xs < x + bandSize; xs++) {
			
				int propagationPixel = y * cols + xs;
				if(tex2D(image, propagationPixel % cols, propagationPixel / cols).x != siteValue) nearestSite[propagationPixel] = pixel;
				else pixel = propagationPixel;
	  
			}

			break;
		}
	
	}

	#pragma unroll
	for(int xp = x + bandSize - 1; xp >= x; xp--) {
	
		int pixel = y * cols + xp;
		if(tex2D(image, pixel % cols, pixel / cols).x == siteValue) {

			#pragma unroll
			for(int xs = xp - 1; xs >= x; xs--) {
			
				int propagationPixel = y * cols + xs;
				float imagePropagationPixel = tex2D(image, propagationPixel % cols, propagationPixel / cols).x;
				int nearestSitePropagationPixel = nearestSite[propagationPixel];
				if(nearestSitePropagationPixel == -1) {
					nearestSite[propagationPixel] = pixel;
				} else if(imagePropagationPixel != siteValue && nearestSitePropagationPixel != -1) {
					float a = abs(nearestSitePropagationPixel % cols - xs);
					float b = abs(xp - xs);
					if(b < a) nearestSite[propagationPixel] = pixel;
				} else pixel = propagationPixel;

			}

			break;
		}

	}
	
}

__global__ void updateBandSitesInRow(int *nearestSite, int cols, int bandSize, int iteration) {

	int globalId = blockIdx.x * blockDim.x + threadIdx.x;
	int x = (globalId / cols) * bandSize + ((iteration + 1) % 2) * (bandSize - 1);
	int y = globalId % cols;
	//int x = blockIdx.x * bandSize + ((iteration + 1) % 2) * (bandSize - 1);
	//int y = threadIdx.x;
	int pixel = y * cols + x;
	int neighbourPixel;

	if(iteration % 2 == 0) neighbourPixel = pixel + 1;
	else neighbourPixel = pixel + bandSize - 1;

	int nearestPixel1 = nearestSite[pixel];
	int nearestPixel2 = nearestSite[neighbourPixel];
	
	float a = GPUComputeEuclideanDistance(pixel, nearestPixel1, cols);
	float b = GPUComputeEuclideanDistance(pixel, nearestPixel2, cols);
	float c = GPUComputeEuclideanDistance(neighbourPixel, nearestPixel1, cols);
	float d = GPUComputeEuclideanDistance(neighbourPixel, nearestPixel2, cols);
	if(a >= b) nearestSite[pixel] = nearestPixel2;
	if(d >= c) nearestSite[neighbourPixel] = nearestPixel1;
	
}

__global__ void updateNearestSiteInRow(int *nearestSite, int cols, int bandSize) {
	
	int pixel = blockIdx.x * blockDim.x + threadIdx.x;
	int x = pixel % cols;
	int y = pixel / cols;
	int band = x / bandSize;
	int firstBandPixel = y * cols + band * bandSize;
	int lastBandPixel = y * cols + band * bandSize + bandSize - 1;
	float a = GPUComputeEuclideanDistance(pixel, nearestSite[pixel], cols);
	float b = GPUComputeEuclideanDistance(pixel, nearestSite[firstBandPixel], cols);
	float c = GPUComputeEuclideanDistance(pixel, nearestSite[lastBandPixel], cols);
	if(b < a && b <= c) nearestSite[pixel] = nearestSite[firstBandPixel];
	if(c < b && c <= a) nearestSite[pixel] = nearestSite[lastBandPixel];

}

__global__ void computeProximateSitesInColumn(int *nearestSite, int *proximateSites, int rows, int cols, int bandSize) {

	//Here, our stack begins in "y + bandSize - 1" and ends in "y"
	int globalId = blockIdx.x * blockDim.x + threadIdx.x;
	int x = globalId % cols;
	int y = (globalId / cols) * bandSize;
	//int x = threadIdx.x;
	//int y = blockIdx.x * bandSize;
	int count = y;

	#pragma unroll
	for(int yb = y; yb < y + bandSize; yb++) {

		int pixel = yb * cols + x;
		int c = nearestSite[pixel];
		if(c != -1) {

			while(count >= y + 2) {
					
				int a = proximateSites[(count - 2) * cols + x];
				int b = proximateSites[(count - 1) * cols + x];
				if(GPUHasDomination(a, b, c, x, cols)) {
					proximateSites[(count - 1) * cols + x] = -1;
					count--;
				} else break;

			}
				
			proximateSites[count * cols + x] = c;
			count++;

		}

	}

}

__global__ void mergeProximateSitesInColumn(int *nearestSite, int *proximateSites, int rows, int cols, int bandSize) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	//int x = threadIdx.x;
	int count = 0;
	
	#pragma unroll
	for(int y = 0; y < bandSize; y++)
		if(proximateSites[y * cols + x] != -1) count++;	
	
	#pragma unroll
	for(int it = 1; it < rows/bandSize; it++) {
		int bandCount = 0;
		
		#pragma unroll
		for(int y = 0; y < bandSize; y++) {

			int yp = y + it * bandSize;
			int pixel = yp * cols + x;
			int c = proximateSites[pixel];
			if(c != -1) {
			
				if(bandCount == 2) {
					proximateSites[count * cols + x] = c;
					count++;
					continue;
				}

				while(count >= 2) {
					
					int a = proximateSites[(count - 2) * cols + x];
					int b = proximateSites[(count - 1) * cols + x];
					if(GPUHasDomination(a, b, c, x, cols)) {
						proximateSites[(count - 1) * cols + x] = -1;
						count--;
						bandCount = 0;
					} else break;

				}
			
				proximateSites[count * cols + x] = c;
				count++;
				bandCount++;
				
			}

		}
	}

}

__global__ void computeNearestSiteInFullKernel(int *proximateSites, int *nearestSite, int rows, int cols) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	//int x = threadIdx.x;
	int count = 0;

	#pragma unroll
	for(int y = 0; y < rows; y++) {

		int pixel = y * cols + x;
		while(count < rows - 1) {
		 
			float a = GPUComputeEuclideanDistance(proximateSites[count * cols + x], pixel, cols);
			float b = GPUComputeEuclideanDistance(proximateSites[(count + 1) * cols + x], pixel, cols);
			if(a <= b) break;
			else count++;
				
				
		}
			
		nearestSite[pixel] = proximateSites[count * cols + x];
			
	}
	
}

template <typename T> __host__ __device__ inline T lerp(T v0, T v1, T t) {
    return (1-t)*v0 + t*v1;
}

__global__ void normalizeDistanceTransform(int *umbraNearestSite, int *litNearestSite, float *normalizedEDTImage, int rows, int cols, float shadowIntensity) {

	int pixel = blockIdx.x * blockDim.x + threadIdx.x;
	float imagePixel = tex2D(image, pixel % cols, pixel / cols).x;
	if(imagePixel != 0.0 && imagePixel != 1.0) {
		float s1 = GPUComputeEuclideanDistance(litNearestSite[pixel], pixel, cols);
		float s2 = GPUComputeEuclideanDistance(umbraNearestSite[pixel], pixel, cols);
		normalizedEDTImage[pixel * 4 + 0] = lerp<float>((s2/(s1 + s2)), 1.0, shadowIntensity);
	} else {
		normalizedEDTImage[pixel * 4 + 0] = lerp<float>(imagePixel, 1.0, shadowIntensity);
	}
	normalizedEDTImage[pixel * 4 + 1] = tex2D(image, pixel % cols, pixel / cols).y;
	
}

void GPUCheckError(char *methodName) {

	hipError_t error = hipGetLastError();
	if(error != hipSuccess) printf("%s: %s\n", methodName, hipGetErrorString(error));
	
}

void GPUClearStructure(int *structure, int rows, int cols, hipStream_t stream) {

    clearStructure<<<(int)ceilf(rows * cols/BLOCK_DIM), BLOCK_DIM, 0, stream>>>(structure);
	GPUCheckError("GPUClearStructure");

}

void GPUClearStructure(int *structure, int rows, int cols) {

    clearStructure<<<(int)ceilf(rows * cols/BLOCK_DIM), BLOCK_DIM>>>(structure);
	GPUCheckError("GPUClearStructure");

}

void GPUComputeNearestSiteInRow(int *nearestSite, float siteValue, int rows, int cols, hipStream_t stream) {

	int bands = 16;
	int bandSize = cols / bands;
	computeSelfNearestSite<<<(int)ceilf(rows * cols/BLOCK_DIM), BLOCK_DIM, 0, stream>>>(nearestSite, siteValue, rows, cols);
	computeNearestSiteInRow<<<(int)ceilf(bands * rows/BLOCK_DIM), BLOCK_DIM, 0, stream>>>(nearestSite, siteValue, rows, cols, bandSize);
	for(int it = 0; it < bands; it++) 
		updateBandSitesInRow<<<(int)ceilf((bands - (int)((it + 1) % 2)) * rows/BLOCK_DIM), BLOCK_DIM, 0, stream>>>(nearestSite, cols, bandSize, it);
	updateNearestSiteInRow<<<(int)ceilf(rows * cols/BLOCK_DIM), BLOCK_DIM, 0, stream>>>(nearestSite, cols, bandSize);
	GPUCheckError("GPUComputeNearestSiteInRow");

}

void GPUComputeProximateSitesInColumn(int *nearestSite, int *proximateSites, int rows, int cols, hipStream_t stream) {

	int bands = 16;
	int bandSize = rows / bands;
	computeProximateSitesInColumn<<<(int)ceilf(bands * cols/BLOCK_DIM), BLOCK_DIM, 0, stream>>>(nearestSite, proximateSites, rows, cols, bandSize);
	mergeProximateSitesInColumn<<<(int)ceilf(cols/BLOCK_DIM), BLOCK_DIM, 0, stream>>>(nearestSite, proximateSites, rows, cols, bandSize);
	GPUCheckError("GPUComputeProximateSitesInColumn");

}

void GPUComputeNearestSiteInFull(int *proximateSites, int *nearestSite, int rows, int cols, hipStream_t stream) {

	computeNearestSiteInFullKernel<<<(int)ceilf(cols/BLOCK_DIM), BLOCK_DIM, 0, stream>>>(proximateSites, nearestSite, rows, cols);
	GPUCheckError("GPUComputeNearestSiteInFull");
	
}

void GPUNormalizeDistanceTransform(int *umbraNearestSite, int *litNearestSite, float *normalizedEDTImage, int rows, int cols, float shadowIntensity) {

	normalizeDistanceTransform<<<(int)ceilf(rows * cols/BLOCK_DIM), BLOCK_DIM>>>(umbraNearestSite, litNearestSite, normalizedEDTImage, rows, cols, shadowIntensity);
	hipMemcpyToArray(arrayDevice2, 0, 0, normalizedEDTImage, rows * cols * 4 * sizeof(float), hipMemcpyDeviceToDevice);
	GPUCheckError("GPUNormalizeDistanceTransform");

}

void GPUCudaBindTexture(hipGraphicsResource_t resource) {

	hipGraphicsMapResources(1, &resource);
	hipGraphicsSubResourceGetMappedArray(&arrayDevice2, resource, 0, 0);
	hipBindTextureToArray(image, arrayDevice2 );
	GPUCheckError("GPUCudaBindTexture");

}

void GPUCudaUnbindTexture(hipGraphicsResource_t resource) {

	hipUnbindTexture(image);
	hipGraphicsUnmapResources(1, &resource);
	GPUCheckError("GPUCudaUnbindTexture");

}